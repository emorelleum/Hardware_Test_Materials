#include "hip/hip_runtime.h"

#include<iostream>
#include<fstream>
#include<string.h>
#include<sstream>//wherever a stringstream pops up, it is to convert a multi-digit number to a string or vice versa
#include"math.h"
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include<time.h>

#include<sys/time.h> // Used for timing this.
#include<unistd.h> // Used for timing this.

#include<algorithm>//needed for next_permutation
#include<climits>//needed for MAX_INT

__device__ __host__ void convertFact(unsigned long long, short*);
__device__ __host__ void orderPermutation(short*, unsigned long long, int);
__device__ __host__ unsigned long long factorial(unsigned long long);

using namespace std;

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
template<class _Ty1, class _Ty2> __device__
	bool prayer(const _Ty1& _Left, const _Ty2& _Right)
{	// test if _Left < _Right and operator< is strict weak ordering
	//	cout <<"!!" << endl;
	if (!(_Left < _Right))
		return (false);
	else if (_Right < _Left)
		;
	//_DEBUG_ERROR2("invalid operator<", _File, _Line);
	return (true);
}

__device__ void swap(short* a, short* b)
{
	//cout <<"swap!!" << endl;
	short temp = *a;
	*a = *b;
	*b = temp;
}

__device__ void reverse(short* a, short* b)
{
	//cout <<"reverse!!" << endl;
	b--;
	while(a < b)
	{
		swap(a,b);
		a++;
		b--;
		//cout << "swapping: " << *a << " with " << *b << endl;
	}
}

__device__ bool nextPerm(short* _First, short* _Last)
{	// permute and test for pure ascending, using operator<
	short* _Next = _Last;
	if (_First == _Last || _First == --_Next)
		return (false);

	for (; ; )
	{	// find rightmost element smaller than successor
		//	cout <<"!!" << endl;
		short* _Next1 = _Next;
		if (prayer(*--_Next, *_Next1))
		{	// swap with rightmost element that's smaller, flip suffix
			short* _Mid = _Last;
			for (; !prayer(*_Next, *--_Mid); )
				;
			swap(_Next, _Mid);
			reverse(_Next1, _Last);
			return (true);
		}

		if (_Next == _First)
		{	// pure descending, flip all
			reverse(_First, _Last);
			return (false);
		}
	}
}
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////




//returns the factorial for the given number
//x: the index for which factorial number you want.
//returns: x!
unsigned long long factorial(unsigned long long x)
{
	if(x == 1)
		return 1;
	else
		return x * factorial(x-1);
}
//converts a number into base factorial
//num: the number to convert to base factorial
//digits: a storage array to store the digits of the base factorial number
//numbers are stored in reverse order (so the 2nd digit is in position 1, the third in 2, etc.
//digits[0] will contain the length of the number in digits, since the first number of a base factorial number is always 0
__device__ __host__ void convertFact(unsigned long long num, short* digits)
{
	int numDigits = 1;//there is always a spare 0 in these numbers
	while(num > 0)
	{
		digits[numDigits] = num % (numDigits + 1);
		numDigits++;
		num /= numDigits;
	}
	digits[0] = numDigits;//because the first digit is always zero, we will store the length of the array in the 0th slot
}
//returns the array transposed to the nth permutation after the given ordering
//toOrder: the set for which you would like to define the nth permutation
//m: the numbered permutation to be stored in toOrder
//size: the size of the array toOrder
//warning: gives unpredictable behavior if n is > the number of unique permutations for toOrder
__device__ __host__ void orderPermutation(short* toOrder, unsigned long long n, int size)
{
	short swaps[100];
	convertFact(n, swaps);
	int location = size - swaps[0];//accounts for leading zeros
	int loc = swaps[0] - 1;//used to iterate through the digits of the factoradic number
	while(loc > 0)
	{
		int temp = toOrder[location + swaps[loc]];
		for(int x = location+swaps[loc]; x > location; x--)//moves all the other numbers up
		{
			toOrder[x] = toOrder[x - 1];
		}
		toOrder[location] = temp;
		location++;
		loc--;
	}

}

__global__ void permute(int* deviceWeights, int* deviceValues, short* deviceItems, long long* devicePermStartIndecies, int* deviceBestValueArray,long long* deviceBestPermArray, int deviceNumItems, int threadsPerBlock)
{


	int id = blockIdx.x * threadsPerBlock + threadIdx.x;

	short* items = deviceItems + (id * deviceNumItems);
	for(int index = 0; index < deviceNumItems; index++)
	{
		items[index] = index;
	}
	
	long long index;

	int totSum;
	int totWeight;

	long long bestPerm = 0;
	int bestValue = 0;

	float sectionRatio = (float)1.0/threadsPerBlock;
	long long block_Perms = devicePermStartIndecies[blockIdx.x + 1] - devicePermStartIndecies[blockIdx.x];
	long long start_Perm = (sectionRatio * threadIdx.x) * block_Perms;
	long long end_Perm = (sectionRatio * (threadIdx.x + 1)) * block_Perms;

	orderPermutation(items, devicePermStartIndecies[blockIdx.x] + start_Perm, deviceNumItems);
	
	int maxWeight = (deviceNumItems * 10)/2;


	for(index = devicePermStartIndecies[blockIdx.x] + start_Perm ; index < devicePermStartIndecies[blockIdx.x] + end_Perm ; index++)
	{
			
		totSum = 0;
		totWeight = 0;
		int i = 0;
		for(int w = 0; w < deviceNumItems; w++)
		{

			totWeight = totWeight + deviceWeights[items[w]];
			if(totWeight > maxWeight)
			{
				break;
			}
			totSum = totSum + deviceValues[items[w]];
				
			i++;
		}

			
		if(totSum > bestValue)
		{
			bestValue = totSum;
			bestPerm = index;
		
		}

		
		nextPerm(items, items + deviceNumItems);
			
		
	}

	deviceBestValueArray[id] = bestValue;
	deviceBestPermArray[id] = bestPerm;



}


int main(int argc, char* argv[])
{

	struct timeval startTime, endTime;
	gettimeofday(&startTime, NULL);

	if(argc != 5)
	{
		cout<<"You have enetered and invalid argument list"<<endl;
		cout<<"Required(number of Items) (seed) (number of blocks) (number of threads per block)"<<endl;
	}

	stringstream arg1(argv[1]);
	int numItems;
	arg1 >> numItems;

	stringstream arg2(argv[2]);
	int randomSeed;
	arg2 >> randomSeed;

	srand(randomSeed);

	
	stringstream arg3(argv[3]);
	int numBlocks;
	arg3 >> numBlocks;


	stringstream arg4(argv[4]);
	int numThreads;
	arg4 >> numThreads;
	

	long long totalThreads = numBlocks * numThreads;


	int maxWeight = (numItems *10) / 2;


	long long* permStartIndecies = (long long*)malloc((numBlocks + 1) * sizeof(long long));

	long long numPerms = factorial(numItems);

	double sectionRatio = double(1)/numBlocks;

	for(int index = 0; index < numBlocks + 1; index++)
	{
		permStartIndecies[index] = numPerms * (sectionRatio * index);
	}
	

	    // take in this round's values and weights
	int* values = (int*)malloc(numItems * sizeof(int));
	int* weights =(int*)malloc(numItems * sizeof(int));
	

	cout<<numItems<<endl;
	cout<<maxWeight<<endl;
	for(int index = 0; index < numItems; index++)
	{
		values[index] = rand() % 30;
		cout<<values[index]<<" ";
	}
	cout<<endl;
	for(int index = 0; index < numItems; index++)
	{
		weights[index] = rand() % 30;
		cout<<weights[index]<<" ";
	}
	cout<<endl;
	
	int* deviceValues;
	int* deviceWeights;
	hipMalloc((void**)&deviceValues, numItems*sizeof(int));
	hipMalloc((void**)&deviceWeights, numItems*sizeof(int));

	hipMemcpy(deviceValues, values, numItems*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWeights, weights, numItems*sizeof(int), hipMemcpyHostToDevice);

	short* deviceItems;
	hipMalloc((void**)&deviceItems, (totalThreads * numItems)*sizeof(short)); 

	long long* devicePermStartIndecies;
	hipMalloc((void**)&devicePermStartIndecies, (numBlocks +1)*sizeof(long long));
	hipMemcpy(devicePermStartIndecies, permStartIndecies, (numBlocks +1)*sizeof(long long), hipMemcpyHostToDevice);
	
	int* deviceBestValueArray;
	long long* deviceBestPermArray;
	hipMalloc((void**)&deviceBestValueArray,(totalThreads) * sizeof(long long));
	hipMalloc((void**)&deviceBestPermArray,(totalThreads) * sizeof(long long));
	


	////entering the parallel section
	permute<<<numBlocks,numThreads>>>(deviceWeights, deviceValues, deviceItems, devicePermStartIndecies, deviceBestValueArray, deviceBestPermArray, numItems, numThreads);


	long long* bestPermArray = (long long*)malloc((totalThreads) * sizeof(long long));
	int* bestValueArray = (int*)malloc((totalThreads) * sizeof(int));

	hipMemcpy(bestPermArray, deviceBestPermArray, (totalThreads) * sizeof(long long), hipMemcpyDeviceToHost);
	hipMemcpy(bestValueArray, deviceBestValueArray, (totalThreads) * sizeof(int), hipMemcpyDeviceToHost);
	

	int finalValue;
	unsigned long long finalPerm = 0;
	int tempValue = 0;
	for(int index = 0; index < totalThreads; index++)
	{
		if(bestValueArray[index] > tempValue)
		{
			tempValue = bestValueArray[index];
			finalPerm = bestPermArray[index];
		}
	}
	finalValue = tempValue;

	short* permArray = new short[numItems];
	for(int index = 0; index < numItems; index++)
	{
		permArray[index] = index;
	}

	orderPermutation(permArray, finalPerm, numItems);
	
	int checkWeight = 0;
	for(int index = 0; index < numItems; index++)
	{
		checkWeight = checkWeight + weights[permArray[index]];
		//cout<<checkWeight<<endl;
		if(checkWeight > maxWeight)
		{
			break;
		}
		cout <<  permArray[index]<< ", ";
	}

	cout<< "best value for theft: "<< finalValue<<endl;
	
	

	gettimeofday(&endTime, NULL);
	long timeDelta, startSeconds, startUSeconds, stopSeconds, stopUSeconds, startTotal, stopTotal;
	startSeconds = startTime.tv_sec;
	stopSeconds = endTime.tv_sec;
	startUSeconds = startTime.tv_usec;
	stopUSeconds = endTime.tv_usec;
	startTotal = (startSeconds * 1000) + (startUSeconds / 1000);
	stopTotal = (stopSeconds * 1000) + (stopUSeconds / 1000);


	timeDelta = stopTotal - startTotal;
	cout << "Time: " << timeDelta << " milliseconds" << endl;
	

	
}
