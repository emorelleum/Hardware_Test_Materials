#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
//#include <windows.h>

using namespace std;

//#include "template_kernel.cu"
//#include <cutil_inline.h>//////////////////////  <---------------------
//#include <template_kernel.cu>

//extern "C"



////////////////////////////////////////////////////////////////////////////////////////////////////////////

//#ifndef _TOTHRAMSEYTEST_KERNEL_H_
//#define _TOTHRAMSEYTEST_KERNEL_H_

//#include <stdio.h>

//const long long MAX_NUM = 100000000;

const long long NUM_GRAPHS = 335544320000LL; // A long long has a maximum value of 9223372036854775807.
const int RED = 0;
const int BLUE = 1;
const int NUM_VERTICES = 45;
const int NUM_EDGES = 990; //change to 990 for 45 vertices


__global__ void Ramsey_Kernel( int *mySourceGraph_device, int *mySolutionGraph_device, bool *foundSolution_device, int N, int numThreads, int *answer_device) 
{
	// Calculate the base graph for this thread.

    //int numThreads; // ASSUME SET BY PARAMETER PASSED TO KERNEL
	
	int offset[NUM_EDGES]; // Holds the binary representation of the offset. The offset is used to calculate the 
                           // first graph in each set of graphs.
	int baseGraph[NUM_EDGES]; // The first graph in the set of graphs to test.
	int graph[NUM_EDGES]; // The graph in each set of graphs to test.
	long long remainder; // The offset from the base graph of the first graph in each range.  Used to calcuate 
                         //the binary value of the offset from the base graph.
	int id = blockIdx.x * blockDim.x + threadIdx.x;  // IDX FROM CUDA...

	long long graphsPerCore = NUM_GRAPHS/numThreads;

	// Calculate the first graph for each core to test.
	
	// Initializing the arrays
	for(int j=0; j < NUM_EDGES; j++)
	{
		baseGraph[j] = 0; // TODO: Make another version of baseGraph to test a different section of graphs
		offset[j] = 0;
		graph[j]=0;
	}//end for

	remainder = graphsPerCore * id;

	// Calculate first graph for each thread to test.
	
	// Translates offset into binary and stores the bits into the offset array
	for(int x = 39; x >= 0; x--)
	{
		long long powResult = pow(2.0f, x);
		if(remainder >= powResult)
		{
			offset[989-x]=1;
			remainder = remainder - powResult;
		}
		else 
		{
			offset[989-x]=0;
		}//end if else
	}//end for


	// Puts sum of offset and base graph into graph array so that
	// graph array holds the bits for the first graph to test.
	for(int m = NUM_EDGES-1; m >= 0; m--)
	{
		graph[m]= graph[m]+offset[m]+baseGraph[m];

		if(graph[m] == 2)
		{
			graph[m]=0;
			graph[m-1] = graph[m-1] +1;
		}
		else if ( graph[m]==3)
		{
			graph[m]= 1;
			graph[m-1] = graph[m-1] +1;
			
		}//end if else
	}//end for

	// Test all the graphs in the range.

	// Tests the first graph.

	//bool result = hasK5(graph);
    //////////////////////////////////////////////////////////////////////////////////////////
    // We replaced the hasK5() function call with the code.
    bool result;

    // Begin hasK5() inlining.
    bool foundK5 = false;

    // initialize counters
    int vertex1 = 0;
    int vertex2 = 1;
    int vertex3 = 2;
    int vertex4 = 3;
    int vertex5 = 4;

    /* These five nested while loops select the 5 vertices that 
       we will test to see if they form a K5. The outermost while loop
       selects the smallest numbered vertex.The next while loop selets the second smallest
       numbered vertex and so on... The set of while loops test every possible
       set of five vertices that could form a K5 until it finds one or has tested every possible
       set.
    */

    while (!foundK5 && vertex1 < NUM_VERTICES - 4)
    {
        while (!foundK5 && vertex2 < NUM_VERTICES - 3)
        {
	        while (!foundK5 && vertex3 < NUM_VERTICES - 2)
	        {
		        while (!foundK5 && vertex4 < NUM_VERTICES - 1)
		        {
			        while (!foundK5 && vertex5 < NUM_VERTICES)
			        {
				        if (graph[((NUM_VERTICES * vertex1) + vertex2 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
					        graph[((NUM_VERTICES * vertex1) + vertex3 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
					        graph[((NUM_VERTICES * vertex1) + vertex4 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
					        graph[((NUM_VERTICES * vertex1) + vertex5 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&							
					        graph[((NUM_VERTICES * vertex2) + vertex3 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
					        graph[((NUM_VERTICES * vertex2) + vertex4 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
					        graph[((NUM_VERTICES * vertex2) + vertex5 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
					        graph[((NUM_VERTICES * vertex3) + vertex4 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == RED &&							
					        graph[((NUM_VERTICES * vertex3) + vertex5 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == RED &&
					        graph[((NUM_VERTICES * vertex4) + vertex5 - (((vertex4 + 1)*(vertex4 + 2))/ 2))] == RED)
				        {
					        foundK5 = true;
				        }
				        else if (graph[((NUM_VERTICES * vertex1) + vertex2 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE&&
					        graph[((NUM_VERTICES * vertex1) + vertex3 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex1) + vertex4 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex1) + vertex5 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex2) + vertex3 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex2) + vertex4 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex2) + vertex5 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex3) + vertex4 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex3) + vertex5 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == BLUE &&
					        graph[((NUM_VERTICES * vertex4) + vertex5 - (((vertex4 + 1)*(vertex4 + 2))/ 2))] == BLUE)
				        {
					        foundK5 = true;
				        }
				        vertex5++;
			        }
			        vertex4++;
			        vertex5 = vertex4 + 1;
		        }
		        vertex3++;
		        vertex4 = vertex3 + 1;
		        vertex5 = vertex4 + 1;
	        }
	        vertex2++;
	        vertex3 = vertex2 + 1;
	        vertex4 = vertex3 + 1;
	        vertex5 = vertex4 + 1;
        }
        vertex1++;
        vertex2 = vertex1 + 1;
        vertex3 = vertex2 + 1;
        vertex4 = vertex3 + 1;
        vertex5 = vertex4 + 1;
    }

    if (foundK5 == true)
    {
        result = true;
    }
    else
    {
        result = false;
    }
    // End hasK5() inlining.
    //////////////////////////////////////////////////////////////////////////////////////////

	// If a graph with no K5 is found, print out that graph.
	if(result == false)
	{
        *foundSolution_device = true;
        answer_device[id] = 1;
	}//end if

	long long loopIndex = 0LL;

	 // This is a for loop that tests all graphs for this core.
	for(loopIndex = 0; loopIndex < graphsPerCore; loopIndex++)
	{
		// Generate next graph.

        // We have replaced the function call for generateNextGraph() with the code.
		//generateNextGraph(graph);
        //////////////////

        bool carry = false;
        int index = NUM_EDGES -1;	

        do 
        {
            //add 1 and check for carry 
            if ( graph[index] == 0)
            {
	            carry = false;
	            graph[index] = 1; //set to 1
            }
            else
            {
	            carry = true;
	            graph[index] = 0; //set to 0
            }//end if/else

            index = index - 1;
        }
        while((carry == true) && (index >= 0));

        index = NUM_EDGES - 1;
        //////////////////


		// Test graph.
        //result = hasK5(graph);

        //////////////////////////////////////////////////////////////////////////////////////////
        // We replaced the hasK5() function call with the code.

        // Begin hasK5() inlining.
        bool foundK5 = false;

        // initialize counters
        int vertex1 = 0;
        int vertex2 = 1;
        int vertex3 = 2;
        int vertex4 = 3;
        int vertex5 = 4;

        /* These five nested while loops select the 5 vertices that 
           we will test to see if they form a K5. The outermost while loop
           selects the smallest numbered vertex.The next while loop selets the second smallest
           numbered vertex and so on... The set of while loops test every possible
           set of five vertices that could form a K5 until it finds one or has tested every possible
           set.
        */

        while (!foundK5 && vertex1 < NUM_VERTICES - 4)
        {
	        while (!foundK5 && vertex2 < NUM_VERTICES - 3)
	        {
		        while (!foundK5 && vertex3 < NUM_VERTICES - 2)
		        {
			        while (!foundK5 && vertex4 < NUM_VERTICES - 1)
			        {
				        while (!foundK5 && vertex5 < NUM_VERTICES)
				        {
					        if (graph[((NUM_VERTICES * vertex1) + vertex2 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
						        graph[((NUM_VERTICES * vertex1) + vertex3 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
						        graph[((NUM_VERTICES * vertex1) + vertex4 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&
						        graph[((NUM_VERTICES * vertex1) + vertex5 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == RED &&							
						        graph[((NUM_VERTICES * vertex2) + vertex3 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
						        graph[((NUM_VERTICES * vertex2) + vertex4 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
						        graph[((NUM_VERTICES * vertex2) + vertex5 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == RED &&							
						        graph[((NUM_VERTICES * vertex3) + vertex4 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == RED &&							
						        graph[((NUM_VERTICES * vertex3) + vertex5 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == RED &&
						        graph[((NUM_VERTICES * vertex4) + vertex5 - (((vertex4 + 1)*(vertex4 + 2))/ 2))] == RED)
					        {
						        foundK5 = true;
					        }
					        else if (graph[((NUM_VERTICES * vertex1) + vertex2 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE&&
						        graph[((NUM_VERTICES * vertex1) + vertex3 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex1) + vertex4 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex1) + vertex5 - (((vertex1 + 1)*(vertex1 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex2) + vertex3 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex2) + vertex4 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex2) + vertex5 - (((vertex2 + 1)*(vertex2 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex3) + vertex4 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex3) + vertex5 - (((vertex3 + 1)*(vertex3 + 2))/ 2))] == BLUE &&
						        graph[((NUM_VERTICES * vertex4) + vertex5 - (((vertex4 + 1)*(vertex4 + 2))/ 2))] == BLUE)
					        {
						        foundK5 = true;
					        }
					        vertex5++;
				        }
				        vertex4++;
				        vertex5 = vertex4 + 1;
			        }
			        vertex3++;
			        vertex4 = vertex3 + 1;
			        vertex5 = vertex4 + 1;
		        }
		        vertex2++;
		        vertex3 = vertex2 + 1;
		        vertex4 = vertex3 + 1;
		        vertex5 = vertex4 + 1;
	        }
	        vertex1++;
	        vertex2 = vertex1 + 1;
	        vertex3 = vertex2 + 1;
	        vertex4 = vertex3 + 1;
	        vertex5 = vertex4 + 1;
        }

        if (foundK5 == true)
        {
	        result = true;
        }
        else
        {
	        result = false;
        }
        // End hasK5() inlining.
        //////////////////////////////////////////////////////////////////////////////////////////

		// If a graph with no K5 is found, print out that graph.
		if(result == false)
		{
            *foundSolution_device = true;
            answer_device[id] = loopIndex+1;
		}//end if
	}//end for loop

}

//#endif


///////////////////////////////////////////////////////////////////////////////////////////////////////////









int cToI(char* c)//used to implement arguments
{
	int length = strlen(c);\
	int ret = 0;
	for(int x = 0; x < length; x++)
	{
		ret *= 10;
		if(c[x] >= '1' && c[x] <= '9')
		{
			ret += c[x] - '1' + 1;
		}
	}
	return ret;
}









int main(int argc, char **argv)
{

	if(argc != 3)
	{
		cout << "Need blocks and Blocks-Per-Thread as arguments" << endl;
	}

	// ********START TIMING********
	time_t start = time(NULL);//starts timer

	int N = 990;
	
	size_t size = N * sizeof(int);

	//*************ALLOCATE MEMORY FOR HOST**********

    // starting graph for GPU to use
	int* mySourceGraph_host = (int*) malloc(size); 

    // place for solution if we find it on host
    int* mySolutionGraph_host = (int*) malloc(size); 

    // flag to tell us if we find a graph
    bool foundSolution_host;

    // This array holds 0 in each slot when initialized.  It will hold the number+1 of
    // the graph that contains no red or blue K5 (if found) after the GPU executes the
    // kernel and the array is copied from the GPU.  In other words, if a slot in the 
    // array holds the value x which is greater than 0, then subtract 1 from x to get
    // the graph (aka 0 to 19,999 or 0 to 9,999) number.  Add that to the (array index *
    // 10,000 or 20,000 depending on how many graphs each thread tests).  Then convert that
    // to binary and voila! you have the graph that shows R(5,5) >= 46.
    int* answer_host = (int*) malloc(16777216 * sizeof(int));

    // Fill the array with zeros.  This function prevents an optimizing compiler from
    // optimizing out this step which can occur with ZeroMemory().
    //SecureZeroMemory(answer_host, sizeof(answer_host));
	for(long long inc = 0; inc < 16777216; inc++)
	{
		answer_host[inc] = 0;
	}

    //***********************************************
	
	cout << "memory allocated on host" << endl <<endl;

    //initialize the memory allocated on host


    for(int i=0; i < N; i++)
    {
        mySourceGraph_host[i] = 0;
        mySolutionGraph_host[i] = -999;
    }
    foundSolution_host = false;

	
	//************ALLOCATE MEMORY ON CUDA CARD********
	int* mySourceGraph_device; 
	hipMalloc(&mySourceGraph_device,size); 

	int* mySolutionGraph_device; 
	hipMalloc(&mySolutionGraph_device,size);

	bool* foundSolution_device; 
	hipMalloc(&foundSolution_device,sizeof(bool));

    int* answer_device;
   // cudaMalloc(&answer_device, (16777216 * sizeof(int)));
hipError_t error_id5 = hipMalloc(&answer_device, (16777216 * sizeof(int)));
	if (error_id5 != hipSuccess) {cout << "OMFG!!!!!!!!!!!" << endl;}
    hipMemset(answer_device, 0, sizeof(*answer_device));
	//************************************************
	
	cout<< "memory allocated on device " <<endl << endl;

 
	//*************TRANSFER FROM HOST TO DEVICE********
	hipMemcpy(mySourceGraph_device, mySourceGraph_host, size, hipMemcpyHostToDevice);

	hipMemcpy(mySolutionGraph_device, mySolutionGraph_host, size, hipMemcpyHostToDevice);

    hipMemcpy(foundSolution_device, &foundSolution_host, sizeof(bool), hipMemcpyHostToDevice);
	//**************************************************

	cout << " transfered memory from host to device " << endl << endl;


    //Determine if this is a CUDA 1.x or 2.x Compute Capability card

    //***********MAJOR.MINOR VERSION NUMBER ***************
	int /*dev,*/ deviceCount = 0;
    int majorNum;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) 
	{
		printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
	} 

		hipSetDevice(0);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		cout<<endl;
		printf("  CUDA Capability Major/Minor version number:   %d.%d\n", deviceProp.major, deviceProp.minor);
		majorNum = deviceProp.major;


/*
	for (dev = 0; dev < deviceCount; ++dev) 
	{
		cudaDeviceProp deviceProp;
		cudaGetDeviceProperties(&deviceProp, dev);
		cout<<endl;
		printf("  CUDA Capability Major/Minor version number:   %d.%d\n", deviceProp.major, deviceProp.minor);
		majorNum = deviceProp.major;
	}//end for loop*/

cout << "The major num is... " << majorNum << endl;
	//*****************************************************

    //*******KERNEL SET-UP***********************
    int numThreads = 0; // The number of threads being used by the GPU.
    //sets the grid,block,thread values depeding on the major and minor numbers
    dim3 grid, block; 
    if (majorNum == 3)//if card is 3.x compute capability, use these parameters for kernel TODO: OMFG UPDATE THIS FOR A REAL SET OF 3.X VALUES!!!
    {
        grid.x = 32768;
	    grid.y = 1;
	    block.x = 1024;
	    block.y = 1;
        numThreads = 33554432;
    }
    else if (majorNum == 2)//if card is 2.x compute capability, use these parameters for kernel
    {
        grid.x = 32768;
	    grid.y = 1;
	    block.x = 1024;
	    block.y = 1;
        numThreads = 33554432;
    }
    else //otherwise if card is 1.x compute capability, use these parameters
    {
        grid.x = 32768;
	    grid.y = 1;
	    block.x = 512;
	    block.y = 1;
        numThreads = 16777216;
    }
	//***********************************************

	//*******KERNEL CALL ****************************
/*    Ramsey_Kernel<<<grid, block>>>(mySourceGraph_device, mySolutionGraph_device, foundSolution_device, N, numThreads, answer_device);
	cout<< " kernel called here " << endl << endl;
*/	//***********************************************
int blocks = cToI(argv[1]);
int threadsPB = cToI(argv[2]);
numThreads = blocks * threadsPB;
//modified Kernel call////////////////////////////
Ramsey_Kernel<<<blocks, threadsPB>>>(mySourceGraph_device, mySolutionGraph_device, foundSolution_device, N, numThreads, answer_device);
	cout<< " kernel called here " << endl << endl;
	
 	
	//*******TRANSFER DATA BACK TO HOST*************
	if (hipMemcpy(&foundSolution_host, foundSolution_device, sizeof(bool)/*size*/, hipMemcpyDeviceToHost) != hipSuccess)
{cout << "FAILLLLLL on copy mem dev-> host" << endl;
	hipError_t error_id2 = hipGetLastError();
cout << "ERROR!!! " << hipGetErrorString(error_id2) << endl;
}
	//cudaMemcpy(&foundSolution_host, foundSolution_device, size, cudaMemcpyDeviceToHost);
	//**********************************************
	
	// Check if a graph with no monochromatic K5 was found and if so
	// print it out.
    if (foundSolution_host == true) 
    {
        cout << "Graph found! " << endl;
        cout << " The graph we found is : " << mySolutionGraph_host << endl;
        // Copy the (64 MB - gack!!!) array holding solution from the GPU to the Host
        // This is only ever done if we actually find the graph that proves R(5,5) >= 46
        // so we never expect it to run and therefore won't get a performance hit...
	    hipMemcpy(answer_host, answer_device, (16777216 * sizeof(int)), hipMemcpyDeviceToHost);
        
        bool foundAnswerInArray = false;
        int counter = 0;
        while ((foundAnswerInArray == false) && (counter < 16777216))
        {
            if (answer_host[counter] > 0) // We found our answer (aka counter-example showing R(5,5) >= 46
            {
                foundAnswerInArray = true;

		        // Calculate the base graph for this thread.

		        int offset[NUM_EDGES]; // Holds the binary representation of the offset. The offset is used to calculate the 
							           // first graph in each set of graphs.
		        int baseGraph[NUM_EDGES]; // The first graph in the set of graphs to test.
		        int graph[NUM_EDGES]; // The graph in each set of graphs to test.
		        long long remainder; // The offset from the base graph of the first graph in each range.  Used to calcuate 
							         //the binary value of the offset from the base graph.

		        int id = counter; // Set id to the first index of the array that holds an answer.
		        int whichGraph = answer_host[counter] - 1; // which graph was it in the thread that found our graph with no K5.

		        long long graphsPerCore = NUM_GRAPHS/numThreads;

		        // Calculate the first graph for each core to test.

		        // Initializing the arrays
		        for(int j=0; j < NUM_EDGES; j++)
		        {
			        baseGraph[j] = 0; // TODO: Make another version of baseGraph to test a different section of graphs
			        offset[j] = 0;
			        graph[j]=0;
		        }//end for

		        remainder = graphsPerCore * id;

		        // Calculate first graph for each thread to test.

		        // Translates offset into binary and stores the bits into the offset array
		        for(int x = 39; x >= 0; x--)
		        {
			        long long powResult = pow(2.0f, x);
			        if(remainder >= powResult)
			        {
				        offset[989-x]=1;
				        remainder = remainder - powResult;
			        }
			        else 
			        {
				        offset[989-x]=0;
			        }//end if else
		        }//end for

		        // Puts sum of offset and base graph into graph array so that
		        // graph array holds the bits for the first graph to test.
		        for(int m = NUM_EDGES-1; m >= 0; m--)
		        {
			        graph[m]= graph[m]+offset[m]+baseGraph[m];

			        if(graph[m] == 2)
			        {
				        graph[m]=0;
				        graph[m-1] = graph[m-1] +1;
			        }
			        else if ( graph[m]==3)
			        {
				        graph[m]= 1;
				        graph[m-1] = graph[m-1] +1;
        				
			        }//end if else
		        }//end for


		        // reinitialize offset array
		        for(int j=0; j < NUM_EDGES; j++)
		        {
			        offset[j] = 0;

		        }//end for


		        // convert whichGraph to binary and shove it in offset[] array
		        // Translates offset into binary and stores the bits into the offset array
		        remainder = whichGraph;
		        for(int x = 39; x >= 0; x--)
		        {
			        long long powResult = pow(2.0f, x);
			        if(remainder >= powResult)
			        {
				        offset[989-x]=1;
				        remainder = remainder - powResult;
			        }
			        else 
			        {
				        offset[989-x]=0;
			        }//end if else
		        }//end for



		        // Puts sum of offset and base graph into graph array so that
		        // graph array holds the bits for the first graph to test.
		        for(int m = NUM_EDGES-1; m >= 0; m--)
		        {
			        graph[m]= graph[m]+offset[m];

			        if(graph[m] == 2)
			        {
				        graph[m]=0;
				        graph[m-1] = graph[m-1] +1;
			        }
			        else if ( graph[m]==3)
			        {
				        graph[m]= 1;
				        graph[m-1] = graph[m-1] +1;
        				
			        }//end if else
		        }//end for

	            //prints out binary graph
	            for(int i = 0; i < NUM_EDGES; i++)
	            {
		            cout << graph[i];

	            }//end for            
            }
            else
            {
                counter ++;
            }
        }
        
    }
    else
    {
        cout << " No Graph was found. " << endl;

    }//end if else


	//******FREE MEMORY ON HOST**********
	//free(mySourceGraph_host);
	//free(mySolutionGraph_host);
	//free(&foundSolution_host);
    //free(answer_host);
	//***********************************

    //******FREE MEMORY ON DEVICE********
	hipFree(mySourceGraph_device);
	hipFree(mySolutionGraph_device);
	hipFree(foundSolution_device);
	hipFree(answer_device);
	//***********************************


	cout << " memory from host and device freed " << endl << endl;

	//*********END TIMING IN SECONDS***************************
	time_t stop = time(NULL);
	int delta = stop - start;
	cout << "Time taken " << delta <<" seconds " << endl;
	//**********************************************
		
}
//end main




